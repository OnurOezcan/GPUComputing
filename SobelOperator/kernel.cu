#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <thread>
#include <chrono>
#include <vector>
#include <time.h>
#include <iostream>
#include <math.h>
#include <omp.h>
#include <filesystem>
#include "imageLoader.h"

#define GRIDVAL 20.0 

//=============================================================================================================================
//                                                  Function Definitions
//=============================================================================================================================
void printCudaDeviceInformation(int maxAvaialbeCores);
void executeSobelOperator(char* image, int maxAvaialbeCores);

//cpu sobel functions (sorted from slowest to fastest)
void separate_step_sobel_cpu_with_indexing(const byte* orig, byte* cpu, const unsigned int width, const unsigned int height, int maxCores);
void separate_step_sobel_cpu_without_indexing(const byte* orig, byte* cpu, const unsigned int width, const unsigned int height, int maxCores);
void combined_step_sobel_cpu(const byte* orig, byte* cpu, const unsigned int width, const unsigned int height);
void sobel_omp(const byte* orig, byte* cpu, const unsigned int width, const unsigned int height);
void test(const byte* image, byte* cpu, const unsigned int width, const unsigned int height, int maxCores);

int imageWidth = 0;

/**
* Index function to access a 1d array like a 2d array
*/
int getIndex(int x, int y) {
    return imageWidth * y + x;
};

/************************************************************************************************
 * void sobel_gpu(const byte*, byte*, uint, uint);
 * - This function runs on the GPU, it works on a 2D grid giving the current x, y pair being worked
 * - on, the const byte* is the original image being processed and the second byte* is the image
 * - being created using the sobel filter. This function runs through a given x, y pair and uses
 * - a sobel filter to find whether or not the current pixel is an edge, the more of an edge it is
 * - the higher the value returned will be
 *
 * Inputs: const byte* orig : the original image being evaluated
 *                byte* cpu : the image being created using the sobel filter
 *               uint width : the width of the image
 *              uint height : the height of the image
 *
 ***********************************************************************************************/
__global__ void sobel_gpu(const byte* orig, byte* cpu, const unsigned int width, const unsigned int height) {
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    float dx, dy;
    if (x > 0 && y > 0 && x < width - 1 && y < height - 1) {
        dx = (-1 * orig[(y - 1) * width + (x - 1)]) + (-2 * orig[y * width + (x - 1)]) + (-1 * orig[(y + 1) * width + (x - 1)]) +
            (orig[(y - 1) * width + (x + 1)]) + (2 * orig[y * width + (x + 1)]) + (orig[(y + 1) * width + (x + 1)]);
        dy = (orig[(y - 1) * width + (x - 1)]) + (2 * orig[(y - 1) * width + x]) + (orig[(y - 1) * width + (x + 1)]) +
            (-1 * orig[(y + 1) * width + (x - 1)]) + (-2 * orig[(y + 1) * width + x]) + (-1 * orig[(y + 1) * width + (x + 1)]);
        cpu[y * width + x] = sqrt((dx * dx) + (dy * dy));
    }
}

/************************************************************************************************
 * int main(int, char*[])
 * - This function is our program's entry point. The function passes in the command line arguments
 * - and if there are exactly 2 command line arguments, the program will continue, otherwise it
 * - will exit with error code 1. If the program continues, it will read in the file given by
 * - command line argument #2 and store as an array of bytes, after some header information is
 * - outputted, the sobel filter will run in 3 different functions on the original image and
 * - 3 new images will be created, each containing a sobel filter created using just the CPU,
 * - OMP, and the GPU, then the image will be written out to a file with an appropriate indicator
 * - appended to the end of the filename.
 *
 * Inputs:    int argc : the number of command line arguments
 *         char*argv[] : an array containing the command line arguments
 * Outputs:   returns 0: code ran successful, no issues came up
 *            returns 1: invalid number of command line arguments
 *            returns 2: unable to process input image
 *            returns 3: unable to write output image
 *
 ***********************************************************************************************/
int main(int argc, char* argv[]) {
    //wraps the input arguments in a vector
    std::vector<char*> arguments(argv, argv + argc);

    //gets the max available number of cpu cores
    int maxAvaialbeCores = std::thread::hardware_concurrency();
    //Check if the user started the program with a valid number of arguments
    if (arguments.size() < 2) {
        printf("%s: Invalid number of command line arguments. Exiting program\n", argv[0]);
        printf("Usage: %s [image.png]", argv[0]);
        return 1;
    }

    //the first argument is no longer needed, therefore it gets removed
    arguments.erase(arguments.begin());

    //print device properties
    printCudaDeviceInformation(maxAvaialbeCores);

    //switch (devProp.major)
    //{
    //case 2: // Fermi
    //    if (devProp.minor == 1) cores *= 48;
    //    else cores *= 32; break;
    //case 3: // Kepler
    //    cores *= 192; break;
    //case 5: // Maxwell
    //    cores *= 128; break;
    //case 6: // Pascal
    //    if (devProp.minor == 1) cores *= 128;
    //    else if (devProp.minor == 0) cores *= 64;
    //    break;
    //}

    //load image (currently only png supported)
    for (char* image : arguments) {
        printf("\n\n########################################################\n");
        printf("#  Starting image processing: %-25.25s#\n", image);
        printf("########################################################\n");
        executeSobelOperator(image, maxAvaialbeCores);
    }

    /** Load our img and allocate space for our modified images **/
    imgData origImg = loadImage(argv[1]);
    imageWidth = origImg.width;
    imgData cpuImg(new byte[origImg.width * origImg.height], origImg.width, origImg.height);
    imgData ompImg(new byte[origImg.width * origImg.height], origImg.width, origImg.height);
    imgData gpuImg(new byte[origImg.width * origImg.height], origImg.width, origImg.height);

    /** make sure all our newly allocated data is set to 0 **/
    memset(cpuImg.pixels, 0, (origImg.width * origImg.height));
    memset(ompImg.pixels, 0, (origImg.width * origImg.height));

    /** We first run the sobel filter on just the CPU using only 1 thread **/
    auto c = std::chrono::system_clock::now();
    combined_step_sobel_cpu(origImg.pixels, cpuImg.pixels, origImg.width, origImg.height);
    std::chrono::duration<double> time_cpu = std::chrono::system_clock::now() - c;

    /** Next, we use OpenMP to parallelize it **/
    c = std::chrono::system_clock::now();
    sobel_omp(origImg.pixels, ompImg.pixels, origImg.width, origImg.height);
    std::chrono::duration<double> time_omp = std::chrono::system_clock::now() - c;

    /** Finally, we use the GPU to parallelize it further **/
    /** Allocate space in the GPU for our original img, new img, and dimensions **/
    byte* gpu_orig, * gpu_sobel;
    hipMalloc((void**)&gpu_orig, (origImg.width * origImg.height));
    hipMalloc((void**)&gpu_sobel, (origImg.width * origImg.height));
    /** Transfer over the memory from host to device and memset the sobel array to 0s **/
    hipMemcpy(gpu_orig, origImg.pixels, (origImg.width * origImg.height), hipMemcpyHostToDevice);
    hipMemset(gpu_sobel, 0, (origImg.width * origImg.height));

    /** set up the dim3's for the gpu to use as arguments (threads per block & num of blocks)**/
    dim3 threadsPerBlock(GRIDVAL, GRIDVAL, 1);
    dim3 numBlocks(ceil(origImg.width / GRIDVAL), ceil(origImg.height / GRIDVAL), 1);

    /** Run the sobel filter using the CPU **/
    c = std::chrono::system_clock::now();
    sobel_gpu <<<numBlocks, threadsPerBlock>>> (gpu_orig, gpu_sobel, origImg.width, origImg.height);
    hipError_t cudaerror = hipDeviceSynchronize(); // waits for completion, returns error code
    if (cudaerror != hipSuccess) fprintf(stderr, "Cuda failed to synchronize: %s\n", hipGetErrorName(cudaerror)); // if error, output error
    std::chrono::duration<double> time_gpu = std::chrono::system_clock::now() - c;
    /** Copy data back to CPU from GPU **/
    hipMemcpy(gpuImg.pixels, gpu_sobel, (origImg.width * origImg.height), hipMemcpyDeviceToHost);

    /** Output runtimes of each method of sobel filtering **/
    printf("\nProcessing %s: %d rows x %d columns\n", argv[1], origImg.height, origImg.width);
    printf("CPU execution time    = %*.1f msec\n", 5, 1000 * time_cpu.count());
    printf("OpenMP execution time = %*.1f msec\n", 5, 1000 * time_omp.count());
    printf("CUDA execution time   = %*.1f msec\n", 5, 1000 * time_gpu.count());
    printf("\nCPU->OMP speedup:%*.1f X", 12, (1000 * time_cpu.count()) / (1000 * time_omp.count()));
    printf("\nOMP->GPU speedup:%*.1f X", 12, (1000 * time_omp.count()) / (1000 * time_gpu.count()));
    printf("\nCPU->GPU speedup:%*.1f X", 12, (1000 * time_cpu.count()) / (1000 * time_gpu.count()));
    printf("\n");

    /** Output the images of each sobel filter with an appropriate string appended to the original image name **/
    writeImage(argv[1], "gpu", gpuImg);
    writeImage(argv[1], "cpu", cpuImg);
    writeImage(argv[1], "omp", ompImg);

    /** Free any memory leftover.. gpuImig, cpuImg, and ompImg get their pixels free'd while writing **/
    hipFree(gpu_orig); hipFree(gpu_sobel);
    return 0;
}

/**
* Output information about the host (CPU) and divce (GPU)
*/
void printCudaDeviceInformation(int maxAvaialbeCores) {
    hipDeviceProp_t cudaDeviceProperties;
    hipGetDeviceProperties(&cudaDeviceProperties, 0);

    printf("########################################################\n");
    printf("#                 Device Information                   #\n");
    printf("########################################################\n");
    printf("CPU: %d Threads\n", std::thread::hardware_concurrency());
    printf("GPU: %s\n\
     CUDA Version %d.%d\n\
     %zd MB global Memory\n\
     %zd KB shared Memory per Block\n\
     %d CUDA cores\n",
        cudaDeviceProperties.name, cudaDeviceProperties.major, cudaDeviceProperties.minor, cudaDeviceProperties.totalGlobalMem >> 20, cudaDeviceProperties.sharedMemPerBlock >> 10, cudaDeviceProperties.multiProcessorCount);
}

void fillExpandedPicture(const byte* originalImage, byte* expandedImage, const unsigned int width, const unsigned int height) {
    //copy data from original image to the expanded image and fill the new added rows/columns
    imageWidth = width;
    for (int y = 0; y < height; y++) {
        for (int x = 0; x < width; x++) {
            //cases for corners
            if (y == 0 && x == 0) {
                expandedImage[getIndex(x, y)] = originalImage[getIndex(x, y)];
            } else if (y == 0 && x == width - 1) {
                expandedImage[getIndex(x, y)] = originalImage[getIndex(x - 2, y)];
            } else if (y == height - 1 && x == 0) {
                expandedImage[getIndex(x, y)] = originalImage[(width - 2) * (y - 2) + x];
            } else if (y == height - 1 && x == width - 1) {
                expandedImage[getIndex(x, y)] = originalImage[(width - 2) * (y - 2) + (x - 2)];
            
            //cases for edges
            } else if (y == 0) {
                expandedImage[getIndex(x, y)] = originalImage[getIndex(x - 1, y)];
            } else if (y > 0 && x == 0) {
                expandedImage[getIndex(x, y)] = originalImage[(width - 2) * (y - 1) + x];
            } else if (y > 0 && x == width - 1) {
                expandedImage[getIndex(x, y)] = originalImage[(width - 2) * (y - 1) + (x - 2)];
            } else if (y == height - 1 && x > 0) {
                expandedImage[getIndex(x, y)] = originalImage[(width - 2) * (y - 2) + (x - 1)];
            }

            //fill in the normal image
            else {
                expandedImage[getIndex(x, y)] = originalImage[(width - 2) * (y - 1) + (x - 1)];
            }
        }
    }
}

void executeSobelOperator(char* image, int maxAvailableCores) {
    //loads the image and allocates memory
    imgData originalImage = loadImage(image);

    //create image that is two wider and two heigher than the original
    imgData expandedImage(new byte[(originalImage.width + 2) * (originalImage.height + 2)], originalImage.width + 2, originalImage.height + 2);
    memset(expandedImage.pixels, 0, (expandedImage.width * expandedImage.height));

    fillExpandedPicture(originalImage.pixels, expandedImage.pixels, expandedImage.width, expandedImage.height);

    //set global image width for index calculation
    imageWidth = originalImage.width;

    //allocate space for the images
    imgData separate_step_cpuImgage(new byte[originalImage.width * originalImage.height], originalImage.width, originalImage.height);
    imgData combined_step_cpuImgage(new byte[originalImage.width * originalImage.height], originalImage.width, originalImage.height); 

    imgData omp_separate_step_cpuImgage(new byte[originalImage.width * originalImage.height], originalImage.width, originalImage.height);
    imgData omp_combined_step_cpuImgage(new byte[originalImage.width * originalImage.height], originalImage.width, originalImage.height);

    imgData omp_combined_step_cpuImgaget(new byte[originalImage.width * originalImage.height], originalImage.width, originalImage.height);

    imgData ompImgage(new byte[originalImage.width * originalImage.height], originalImage.width, originalImage.height);
    imgData gpuImgage(new byte[originalImage.width * originalImage.height], originalImage.width, originalImage.height);

    memset(separate_step_cpuImgage.pixels, 0, (originalImage.width * originalImage.height));
    memset(combined_step_cpuImgage.pixels, 0, (originalImage.width * originalImage.height));
    memset(omp_separate_step_cpuImgage.pixels, 0, (originalImage.width * originalImage.height));
    memset(omp_combined_step_cpuImgage.pixels, 0, (originalImage.width * originalImage.height));

    memset(omp_combined_step_cpuImgaget.pixels, 0, (originalImage.width * originalImage.height));

    //definitions for time measurement
    std::chrono::system_clock::time_point start;
    std::chrono::duration<double> serparatedStepWithIndexingCPUTime;
    std::chrono::duration<double> serparatedStepWithoutIndexingCPUTime;

    std::chrono::duration<double> serparatedStepWithIndexingOMPTime;
    std::chrono::duration<double> serparatedStepWithoutIndexingOMPTime;


    std::chrono::duration<double> serparatedStepWithoutIndexingOMPTimetttt;

    //Single core sobel function with indexing and seperated steps (3 total steps) 
    start = std::chrono::system_clock::now();
    separate_step_sobel_cpu_with_indexing(expandedImage.pixels, separate_step_cpuImgage.pixels, expandedImage.width, expandedImage.height, 1);
    serparatedStepWithIndexingCPUTime = std::chrono::system_clock::now() - start;

    ////Single core soble function without indexing and seperated steps (3 total steps) 
    //start = std::chrono::system_clock::now();
    //separate_step_sobel_cpu_without_indexing(expandedImage.pixels, combined_step_cpuImgage.pixels, expandedImage.width, expandedImage.height, 1);
    //serparatedStepWithoutIndexingCPUTime = std::chrono::system_clock::now() - start;

    //Multi core sobel function with indexing and seperated steps (3 total steps) 
    start = std::chrono::system_clock::now();
    separate_step_sobel_cpu_with_indexing(expandedImage.pixels, omp_separate_step_cpuImgage.pixels, expandedImage.width, expandedImage.height, maxAvailableCores);
    serparatedStepWithIndexingOMPTime = std::chrono::system_clock::now() - start;

    ////Multi core soble function without indexing and seperated steps (3 total steps) 
    //start = std::chrono::system_clock::now();
    //separate_step_sobel_cpu_without_indexing(expandedImage.pixels, omp_combined_step_cpuImgage.pixels, expandedImage.width, expandedImage.height, maxAvailableCores);
    //serparatedStepWithoutIndexingOMPTime = std::chrono::system_clock::now() - start;

    //test
    /*start = std::chrono::system_clock::now();
    test(originalImage.pixels, omp_combined_step_cpuImgaget.pixels, originalImage.width, originalImage.height, 1);
    serparatedStepWithoutIndexingOMPTimetttt = std::chrono::system_clock::now() - start;*/



    printf("CPU execution time    = %*.1f msec\n", 5, 1000 * serparatedStepWithIndexingCPUTime.count());
    printf("CPU execution time    = %*.1f msec\n", 5, 1000 * serparatedStepWithoutIndexingCPUTime.count());
    printf("CPU execution time    = %*.1f msec\n", 5, 1000 * serparatedStepWithIndexingOMPTime.count());
    printf("CPU execution time    = %*.1f msec\n", 5, 1000 * serparatedStepWithoutIndexingOMPTime.count());
    //printf("CPU execution time    = %*.1f msec\n", 5, 1000 * serparatedStepWithoutIndexingOMPTimetttt.count());

    writeImage(image, "cpuu", separate_step_cpuImgage);
    writeImage(image, "cpuut", combined_step_cpuImgage);
}

void separate_step_sobel_cpu_with_indexing(const byte* image, byte* cpu, const unsigned int width, const unsigned int height, int maxCores) {
    int* dx = new int[(width - 2) * (height - 2)];
    int* dy = new int[(width - 2) * (height - 2)];

    omp_set_num_threads(maxCores);
    imageWidth = width;
    #pragma omp parallel for
    for (int y = 1; y < height - 1; y++) {
        for (int x = 1; x < width - 1; x++) {
            dx[(width - 2) * (y - 1) + (x - 1)] = (1 * image[getIndex(x - 1, y - 1)]) + (-1 * image[getIndex(x + 1, y - 1)]) +
                (2 * image[getIndex(x - 1, y)]) + (-2 * image[getIndex(x + 1, y)]) +
                (1 * image[getIndex(x - 1, y + 1)]) + (-1 * image[getIndex(x + 1, y + 1)]);
        }
    }

    #pragma omp parallel for
    for (int y = 1; y < height - 1; y++) {
        for (int x = 1; x < width - 1; x++) {
            dy[(width - 2) * (y - 1) + (x - 1)] = (1 * image[getIndex(x - 1, y - 1)]) + (2 * image[getIndex(x, y - 1)]) + (1 * image[getIndex(x + 1, y - 1)]) +
                (-1 * image[getIndex(x - 1, y + 1)]) + (-2 * image[getIndex(x, y + 1)]) + (-1 * image[getIndex(x + 1, y + 1)]);
        }
    }

    imageWidth = width - 2;
    #pragma omp parallel for
    for (int y = 1; y < height - 1; y++) {
        for (int x = 1; x < width - 1; x++) {
            cpu[getIndex(x - 1, y - 1)] = sqrt((dx[getIndex(x - 1, y - 1)] * dx[getIndex(x - 1, y - 1)]) + (dy[getIndex(x - 1, y - 1)] * dy[getIndex(x - 1, y - 1)]));
        }
    }
}

void separate_step_sobel_cpu_without_indexing(const byte* image, byte* cpu, const unsigned int width, const unsigned int height, int maxCores) {
    int* dx = new int[width * height];
    int* dy = new int[width * height];

    omp_set_num_threads(maxCores);

    #pragma omp parallel for
    for (int y = 1; y < height - 1; y++) {
        for (int x = 1; x < width - 1; x++) {
            dx[y * width + x] = (1 * image[width * (y - 1) + (x - 1)]) + (-1 * image[width * (y - 1) + (x + 1)]) +
                (2 * image[width * y + (x - 1)]) + (-2 * image[width * y + (x + 1)]) +
                (1 * image[width * (y + 1) + (x - 1)]) + (-1 * image[width * (y + 1) + (x + 1)]);
        }
    }

    #pragma omp parallel for
    for (int y = 1; y < height - 1; y++) {
        for (int x = 1; x < width - 1; x++) {
            dy[y * width + x] = (image[(y - 1) * width + (x - 1)]) + (2 * image[(y - 1) * width + x]) + (image[(y - 1) * width + (x + 1)]) +
                (-1 * image[(y + 1) * width + (x - 1)]) + (-2 * image[(y + 1) * width + x]) + (-1 * image[(y + 1) * width + (x + 1)]);
        }
    }

    #pragma omp parallel for
    for (int y = 1; y < height - 1; y++) {
        for (int x = 1; x < width - 1; x++) {
            cpu[y * width + x] = sqrt((dx[y * width + x] * dx[y * width + x]) + (dy[y * width + x] * dy[y * width + x]));
        }
    }
}

void test(const byte* image, byte* cpu, const unsigned int width, const unsigned int height, int maxCores) {
    int* dx = new int[width * height];
    int* dy = new int[width * height];

    omp_set_num_threads(maxCores);

    #pragma omp parallel for
    for (int y = 1; y < height - 1; y++) {
        for (int x = 1; x < width - 1; x++) {
            dx[y * width + x] = (1 * image[width * (y - 1) + (x - 1)]) + (-1 * image[width * (y - 1) + (x + 1)]) +
                (2 * image[width * y + (x - 1)]) + (-2 * image[width * y + (x + 1)]) +
                (1 * image[width * (y + 1) + (x - 1)]) + (-1 * image[width * (y + 1) + (x + 1)]);
        }
    }

    #pragma omp parallel for
    for (int y = 1; y < height - 1; y++) {
        for (int x = 1; x < width - 1; x++) {
            dy[y * width + x] = (image[(y - 1) * width + (x - 1)]) + (image[(y - 1) * width + x] << 1) + (image[(y - 1) * width + (x + 1)]) +
                (-1 * image[(y + 1) * width + (x - 1)]) + (-image[(y + 1) * width + x] << 1) + (-1 * image[(y + 1) * width + (x + 1)]);
        }
    }

    #pragma omp parallel for
    for (int y = 1; y < height - 1; y++) {
        for (int x = 1; x < width - 1; x++) {
            cpu[y * width + x] = sqrt((dx[y * width + x] * dx[y * width + x]) + (dy[y * width + x] * dy[y * width + x]));
        }
    }
}

/************************************************************************************************
 * void combined_step_sobel_cpu(const byte*, byte*, uint, uint);
 * - This function runs on just the CPU with nothing running in parallel. The function takes in
 * - an original image and compares the pixels to the left and right and then above and below
 * - to find the rate of change of the two comparisons, then squares, adds, and square roots the
 * - pair to find a 'sobel' value, this value is saved into an array of bytes and then loops to
 * - handle the next pixel. The resulting array of evaluated pixels should be of an image showing
 * - in black and white where edges appear in the original image.
 *
 * Inputs: const byte* orig : the original image being evaluated
 *                byte* cpu : the image being created using the sobel filter
 *               uint width : the width of the image
 *              uint height : the height of the image
 *
 ***********************************************************************************************/
void combined_step_sobel_cpu(const byte* orig, byte* cpu, const unsigned int width, const unsigned int height) {
    /*for (int y = 1; y < height - 1; y++) {
        for (int x = 1; x < width - 1; x++) {
            int dx = (-1 * orig[(y - 1) * width + (x - 1)]) + (-2 * orig[y * width + (x - 1)]) + (-1 * orig[(y + 1) * width + (x - 1)]) +
                (orig[(y - 1) * width + (x + 1)]) + (2 * orig[y * width + (x + 1)]) + (orig[(y + 1) * width + (x + 1)]);
            int dy = (orig[(y - 1) * width + (x - 1)]) + (2 * orig[(y - 1) * width + x]) + (orig[(y - 1) * width + (x + 1)]) +
                (-1 * orig[(y + 1) * width + (x - 1)]) + (-2 * orig[(y + 1) * width + x]) + (-1 * orig[(y + 1) * width + (x + 1)]);
            cpu[y * width + x] = sqrt((dx * dx) + (dy * dy));
        }
    }*/

    /*for (int y = 1; y < height - 1; y++) {
        for (int x = 1; x < width - 1; x++) {
            int dx = (1 * orig[getIndex(x - 1, y - 1)]) + (-1 * orig[getIndex(x + 1, y - 1)]) +
                (2 * orig[getIndex(x - 1, y)]) + (-2 * orig[getIndex(x + 1, y)]) +
                (1 * orig[getIndex(x - 1, y + 1)]) + (-1 * orig[getIndex(x + 1, y + 1)]);

            int dy = (1 * orig[getIndex(x - 1, y - 1)]) + (2 * orig[getIndex(x, y - 1)]) + (1 * orig[getIndex(x + 1, y - 1)]) +
                (-1 * orig[getIndex(x - 1, y + 1)]) + (-2 * orig[getIndex(x, y + 1)]) + (-1 * orig[getIndex(x + 1, y + 1)]);

            cpu[getIndex(x, y)] = sqrt((dx * dx) + (dy * dy));
        }
    }*/
    //imageWidth* y + x;
    for (int y = 1; y < height - 1; y++) {
        for (int x = 1; x < width - 1; x++) {
            int dx = (1 * orig[imageWidth * (y - 1) + (x - 1)]) + (-1 * orig[imageWidth * (y - 1) + (x + 1)]) +
                (2 * orig[imageWidth * y + (x - 1)]) + (-2 * orig[imageWidth * y + (x + 1)]) +
                (1 * orig[imageWidth * (y + 1) + (x - 1)]) + (-1 * orig[imageWidth * (y + 1) + (x + 1)]);

            int dy = (1 * orig[imageWidth * (y - 1) + (x - 1)]) + (2 * orig[imageWidth * (y - 1) + x]) + (1 * orig[imageWidth * (y - 1) + (x + 1)]) +
                (-1 * orig[imageWidth * (y + 1) + (x - 1)]) + (-2 * orig[imageWidth * (y + 1) + x]) + (-1 * orig[imageWidth * (y + 1) + (x + 1)]);

            cpu[imageWidth * y + x] = sqrt((dx * dx) + (dy * dy));
        }
    }
}


/************************************************************************************************
 * void sobel_omp(const byte*, byte*, uint, uint);
 * - This function runs on the CPU but uses OpenMP to parallelize the for workload. The function
 * - is identical to the sobel_cpu function in what it does, except there is a #pragma call for
 * - the compiler to seperate out the for loop across different cores. Each pixel is able to be
 * - worked on independantly of all other pixels, so there is no worry of one thread messing up
 * - another thread. The resulting array is the same as the cpu function, producing an image in
 * - black and white of where edges appear in the original image.
 *
 * Inputs: const byte* orig : the original image being evaluated
 *                byte* cpu : the image being created using the sobel filter
 *               uint width : the width of the image
 *              uint height : the height of the image
 *
 ***********************************************************************************************/
void sobel_omp(const byte* orig, byte* cpu, const unsigned int width, const unsigned int height) {
#pragma omp parallel for
    for (int y = 1; y < height - 1; y++) {
        for (int x = 1; x < width - 1; x++) {
            int dx = (-1 * orig[(y - 1) * width + (x - 1)]) + (-2 * orig[y * width + (x - 1)]) + (-1 * orig[(y + 1) * width + (x - 1)]) +
                (orig[(y - 1) * width + (x + 1)]) + (2 * orig[y * width + (x + 1)]) + (orig[(y + 1) * width + (x + 1)]);
            int dy = (orig[(y - 1) * width + (x - 1)]) + (2 * orig[(y - 1) * width + x]) + (orig[(y - 1) * width + (x + 1)]) +
                (-1 * orig[(y + 1) * width + (x - 1)]) + (-2 * orig[(y + 1) * width + x]) + (-1 * orig[(y + 1) * width + (x + 1)]);
            cpu[y * width + x] = sqrt((dx * dx) + (dy * dy));
        }
    }
}

