#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <math.h>      
#include <stdio.h>
#include <math.h>
#include <time.h> 
#include <omp.h>
#include <hip/hip_cooperative_groups.h>

// A routine to give access to a high precision timer on most systems.
#if defined(_WIN32)
#if !defined(WIN32_LEAN_AND_MEAN)
#define WIN32_LEAN_AND_MEAN
#endif
#include <windows.h>
double second(void)
{
	LARGE_INTEGER t;
	static double oofreq;
	static int checkedForHighResTimer;
	static BOOL hasHighResTimer;

	if (!checkedForHighResTimer) {
		hasHighResTimer = QueryPerformanceFrequency(&t);
		oofreq = 1.0 / (double)t.QuadPart;
		checkedForHighResTimer = 1;
	}
	if (hasHighResTimer) {
		QueryPerformanceCounter(&t);
		return (double)t.QuadPart * oofreq;
	}
	else {
		return (double)GetTickCount() * 1.0e-3;
	}
}
#elif defined(__linux__) || defined(__APPLE__)
#include <stddef.h>
#include <sys/time.h>
double second(void)
{
	struct timeval tv;
	gettimeofday(&tv, NULL);
	return (double)tv.tv_sec + (double)tv.tv_usec * 1.0e-6;
}
#else
#error unsupported platform
#endif

void exercise1();
void exercise3();

int main() {
	//init random seed
	srand((unsigned) second());

	//exercise1();
	exercise3();
}


void cudaCheckError() {
	hipError_t e = hipGetLastError();                                 
	if (e != hipSuccess) {
		printf("Cuda failure %s:%d: '%s'\n", __FILE__, __LINE__, hipGetErrorString(e));
		exit(0); 
	}
}

//=====================================================================================
//									Aufgabe 1										 //
//=====================================================================================
const int N = pow(10, 8);
#define BLOCK_SIZE 5000;
#define TOTAL_NUMBER_OF_THREADS 20000;

__global__ void addValuesA(int* matrix, int* interimResults) {
	const int i = blockIdx.x * blockDim.x + threadIdx.x;
	const int blockSize = BLOCK_SIZE;
	const int totalNumberOfThreads = TOTAL_NUMBER_OF_THREADS;

	const int max = blockSize * (i + 1) - 1;
	int index = blockSize * i;
	long long added = 0;

	for (; index <= max; index++) {
		added += matrix[index];
	}
	interimResults[i] = added;
}

__global__ void addValuesB(int* matrix, int* interimResults) {
	const int i = blockIdx.x * blockDim.x + threadIdx.x;
	const int blockSize = BLOCK_SIZE;
	const int totalNumberOfThreads = TOTAL_NUMBER_OF_THREADS;

	long long added = 0;
	for (int index = 0; index < blockSize; index++) {
		added += matrix[i + index * totalNumberOfThreads];
	}
	interimResults[i] = added;
}


void exercise1() {
	/*
		A is slightly faster than B due to the memory management. This is because the data is always loaded in blocks. Since A calculates the data linearly, fewer data blocks have to be loaded by the CPU
	*/
	const int numberOfBlocks = 32;
	const int numberOfThreads = 625;
	const int totalNumberOfThreads = TOTAL_NUMBER_OF_THREADS;

	printf("Start of Exercise 1!\n");
	int* matrix;
	int* interimResultsA;
	int* interimResultsB;
	long long resultA = 0;
	long long resultB = 0;

	//allocate global memory
	hipMallocManaged(&matrix, N * sizeof(int)); 
	cudaCheckError();

	hipMallocManaged(&interimResultsA, totalNumberOfThreads * sizeof(int));
	cudaCheckError(); 
	
	hipMallocManaged(&interimResultsB, totalNumberOfThreads * sizeof(int));
	cudaCheckError();

	double timeStart, timeStop, timeLinear;
	double timeStartA, timeStopA, timeParallelA;
	double timeStartB, timeStopB, timeParallelB;

	long long added = 0;
	timeStart = second();
	for (unsigned int i = 0; i < N; i++) {
		matrix[i] = rand() % 10;
		added += matrix[i];
	}

	timeStop = second();

	printf("Linear Calculated Checksum: %d\n", added);
	timeLinear = timeStop - timeStart;
	printf("Time: %f\n\n", timeLinear);

	//b)
	timeStartB = second();
	addValuesB << <numberOfBlocks, numberOfThreads >> > (matrix, interimResultsB);
	cudaCheckError();
	hipDeviceSynchronize();
	timeStopB = second();
	timeParallelB = timeStopB - timeStartB;

	for (int i = 0; i < totalNumberOfThreads; i++) {
		resultB += interimResultsB[i];
	}
	printf("Sum of b): %d\n", resultB);
	printf("Time A: %f\n\n", timeParallelB);

	//a)
	timeStartA = second();
	addValuesA << <numberOfBlocks, numberOfThreads >> > (matrix, interimResultsA);
	cudaCheckError();
	hipDeviceSynchronize();
	timeStopA = second();
	timeParallelA = timeStopA - timeStartA;

	for (int i = 0; i < totalNumberOfThreads; i++) {
		resultA += interimResultsA[i];
	}
	printf("Sum of a): %d\n", resultA);
	printf("Time A: %f\n\n", timeParallelA);

	hipFree(matrix);
	cudaCheckError();

	hipFree(interimResultsA);
	cudaCheckError();

	hipFree(interimResultsB);
	cudaCheckError();
}

//=====================================================================================
//									Aufgabe 2										 //
//=====================================================================================

/*
	NVIDIA RTX 2080 (8GB Memory)
	Every calculation was performed 10 times and the average is found in the table below


	Dimension |	CPU				| GPU			 | factor
	-----------------------------------------------------
	10		  | 0.000001		| 0.000168		 | 0,006
	100		  | 0.000012		| 0.000311		 | 0,038
	1.000	  | 0.001100		| 0.002683		 | 0,409
	10.000	  | 0.106819		| 0.141655       | 0,75
	20.000	  | 0.422229		| 0.520808		 | 0,81
	30.000	  | 0.955817	    | 1.192795		 | 0,80

	As we can see in the table is the CPU always fast in this specifi calculation than the GPU
	But as the number of calculations get higher the CPU and GPU get closer with the time it takes to calculate. So its possible that the GPU is faster than the CPU with a bigger matrix. But due to memory limitations i cant test this.
*/

const int DIMENSION = 30000;
const int NUMBER_OF_BLOCKS = 100;
const int NUMBER_OF_THREADS = DIMENSION / NUMBER_OF_BLOCKS;

__global__ void calculateMatrix(unsigned int dimension, int* matrix, int* vector, long long* result) {
	int threadId = blockIdx.x * blockDim.x + threadIdx.x;
	const unsigned int size = dimension * (dimension + 1) / 2;
	long index = size - ((dimension - threadId) * (dimension - threadId + 1) / 2);

	for (unsigned int j = 0; j < dimension - threadId; j++) {
		result[threadId] += matrix[index + j] * vector[j];
	}
}

void exercise3() {
	printf("Start of Exercise 3!\n");

	//measures time to initialize matrix and vectors
	double timeStartInitMatrix, timeStopInitMatrix, timeToInitMatrix;
	double timeStartInitVector, timeStopInitVector, timeToInitVector;
	double timeStartCalculate, timeStopCalculate, timeToCalculate;

	//initializes the matrix
	int* matrix;

	int size = (DIMENSION * (DIMENSION + 1)) / 2;
	timeStartInitMatrix = second();

	hipMallocManaged(&matrix, size * sizeof(int));
	cudaCheckError();

	for (int i = 0; i < size; i++) {
		matrix[i] = rand() % 50;
	}

	timeStopInitMatrix = second();
	timeToInitMatrix = timeStopInitMatrix - timeStartInitMatrix;
	printf("Time to init Matrix: %f\n", timeToInitMatrix);

	//initializes the vector
	int* vector;
	timeStartInitVector = second();

	hipMallocManaged(&vector, DIMENSION * sizeof(int));
	cudaCheckError();

	for (int i = 0; i < DIMENSION; i++) {
		vector[i] = rand() % 50;
	}

	timeStopInitVector = second();
	timeToInitVector = timeStopInitVector - timeStartInitVector;
	printf("Time to init Vector: %f\n", timeToInitVector);

	//initialize result vector
	long long* result;

	hipMallocManaged(&result, DIMENSION * sizeof(long long));
	cudaCheckError();

	//calculate result
	timeStartCalculate = second();
	calculateMatrix << <NUMBER_OF_BLOCKS, NUMBER_OF_THREADS >> > (DIMENSION, matrix, vector, result);
	cudaCheckError();
	hipDeviceSynchronize();
	timeStopCalculate = second();
	timeToCalculate = timeStopCalculate - timeStartCalculate;
	printf("Time to Calculate: %f\n", timeToCalculate);

	hipFree(matrix);
	hipFree(vector);
	hipFree(result);
}

//=====================================================================================
//									Aufgabe 5										 //
//=====================================================================================
